#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <sstream>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>

#include <dlfcn.h>
#include <unistd.h>



#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    std::cerr << "CUDA_CHECK: " << hipGetErrorString(code) << " " << file << " "
        << line << std::endl;
    if (abort)
      exit(code);
  }
}

#define DR_CHECK(ans)                                                   \
  { rtAssert((ans), __FILE__, __LINE__); }
inline void rtAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    const char *str;
    hipDrvGetErrorString(code, &str);
    std::cerr << "DR_CHECK: " << str << " " << file << " "
        << line << std::endl;
    if (abort)
      exit(code);
  }
}

#define DH_BODY DR_CHECK(hipMemcpyDtoH(h, (uintptr_t)d, n * sizeof(float)))
#define HD_BODY DR_CHECK(hipMemcpyHtoD((uintptr_t)d, h, n * sizeof(float)))

void dh1(float *h, const float*d, const size_t n) {
  DH_BODY;
}
void dh2(float *h, const float*d, const size_t n) {
  DH_BODY;
}

void hd1(float *d, const float*h, const size_t n) {
  HD_BODY;
}
void hd2(float *d, const float*h, const size_t n) {
  HD_BODY;
}
#undef DH_BODY
#undef HD_BODY

const size_t N = 8 * 1024 * 1024;

void touch(float *f, const size_t e, const size_t n) {
  const size_t stride = std::max(1ul, e / sizeof(float));
  for (int i = 0; i < n; i += stride) {
    f[i] = rand();
  }
}


int main(void) {

  const long pageSize = sysconf(_SC_PAGESIZE);

  int numDevices;
  CUDA_CHECK(hipGetDeviceCount(&numDevices));
  fprintf(stderr, "%d devices\n", numDevices);

  // set up host allocations
  float *hpn;
  CUDA_CHECK(hipHostMalloc(&hpn, N * sizeof(float)));

  // setup device allocations
  float **d = new float*[numDevices];
  for (int i = 0; i < numDevices; ++i) {
    d[i] = nullptr;
    CUDA_CHECK(hipMalloc(&d[i], N * sizeof(float)));
  }

  // Touch host memory
  touch(hpn, pageSize, N);

    // memcpy sequence
  for (int i = 0; i < numDevices; ++i) {
    std::stringstream buffer;
    buffer << "pn cpu->" << i;
    nvtxRangePush(buffer.str().c_str());
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyHtoD((uintptr_t)d[i], hpn, N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyHtoD((uintptr_t)d[i], hpn, N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyHtoD((uintptr_t)d[i], hpn, N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyHtoD((uintptr_t)d[i], hpn, N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    touch(hpn, pageSize, N);
    hd1(d[i], hpn, N);
    touch(hpn, pageSize, N);
    hd2(d[i], hpn, N);
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    touch(hpn, pageSize, N);
    DR_CHECK(hipMemcpyDtoH(hpn, (uintptr_t)d[i], N * sizeof(float)));
    nvtxRangePop();
  }

    CUDA_CHECK(hipHostFree(hpn));
    return 0;
}
