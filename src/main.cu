#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    std::cerr << "CUDA_CHECK: " << hipGetErrorString(code) << " " << file << " "
        << line << std::endl;
    if (abort)
      exit(code);
  }
}

int main(void) {
    float *h = new float[1000];
    float *d = nullptr;


    CUDA_CHECK(hipMalloc(&d, 1000 * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d, h, 1000 * sizeof(float), hipMemcpyHostToDevice));


    delete[] h;
    CUDA_CHECK(hipFree(d));
    return 0;
}